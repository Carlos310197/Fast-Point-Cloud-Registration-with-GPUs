#include "hip/hip_runtime.h"
//For debugging:
//nvcc ICP_standard.cu -lcublas -lcurand -lcusolver -o ICP_cuda

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>
#include <hiprand.h>
#include <hipsolver.h>
//#include <hip/device_functions.h>

//constants
#define WIDTH 32
#define NUM_POINTS WIDTH*WIDTH //width of grid (1024 points)
#define XY_min -2.0
#define XY_max 2.0
#define MAX_ITER 40

void SmatrixMul(float* A, float* B, float* C, int m, int n, int k);
void printScloud(float* cloud, int num_points, int points2show);
void printSarray(float* array, int points2show);
void printIarray(int* array, int points2show);

__global__
void Matching(float* P, float* Q, int q_points, int* idx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float min = 100000;
	float d;
	for (int j = 0; j < q_points; j++)
	{
		d = (float)sqrt((P[0 + i * 3] - Q[0 + j * 3]) * (P[0 + i * 3] - Q[0 + j * 3]) +
			(P[1 + i * 3] - Q[1 + j * 3]) * (P[1 + i * 3] - Q[1 + j * 3]) +
			(P[2 + i * 3] - Q[2 + j * 3]) * (P[2 + i * 3] - Q[2 + j * 3]));
		if (d < min)
		{
			min = d;
			idx[i] = j;
		}
	}
}

__global__
void Q_index(float* Q, int* idx, float* Q_idx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	Q_idx[0 + i * 3] = Q[0 + idx[i] * 3];
	Q_idx[1 + i * 3] = Q[1 + idx[i] * 3];
	Q_idx[2 + i * 3] = Q[2 + idx[i] * 3];
}

__global__
void deviation(float* P, float* Q, float* barP, float* barQ, float* devP, float* devQ)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	devP[0 + i * 3] = P[0 + i * 3] - barP[0];
	devP[1 + i * 3] = P[1 + i * 3] - barP[1];
	devP[2 + i * 3] = P[2 + i * 3] - barP[2];

	devQ[0 + i * 3] = Q[0 + i * 3] - barQ[0];
	devQ[1 + i * 3] = Q[1 + i * 3] - barQ[1];
	devQ[2 + i * 3] = Q[2 + i * 3] - barQ[2];
}

__global__
void RyT(float* R, float* T, float* P, float* Q)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	Q[0 + i * 3] = R[0 + 0 * 3] * P[0 + i * 3] + R[0 + 1 * 3] * P[1 + i * 3] + R[0 + 2 * 3] * P[2 + i * 3] + T[0];
	Q[1 + i * 3] = R[1 + 0 * 3] * P[0 + i * 3] + R[1 + 1 * 3] * P[1 + i * 3] + R[1 + 2 * 3] * P[2 + i * 3] + T[1];
	Q[2 + i * 3] = R[2 + 0 * 3] * P[0 + i * 3] + R[2 + 1 * 3] * P[1 + i * 3] + R[2 + 2 * 3] * P[2 + i * 3] + T[2];
}

int main()
{
	int num_points, i, j, k;
	float ti[3], ri[3];
	float lin_space[WIDTH], lenght;

	num_points = WIDTH * WIDTH;//number of points
	lenght = XY_max - XY_min;

	//for this specific case the number of points of the 2 clouds are the same
	int d_points = num_points;
	int m_points = num_points;

	////////////////1st:Creation of the synthetic data//////////////

	//create an array with all points equally separated
	int n = WIDTH;
	for (i = 0; i < WIDTH; i++)
	{
		lin_space[i] = (float)XY_min + ((float)i * (float)lenght) / (float(n) - 1.0f);
	}

	//create the meshgrid
	float* mesh_x = (float*)malloc(num_points * sizeof(float));
	float* mesh_y = (float*)malloc(num_points * sizeof(float));

	if ((mesh_x != NULL) && (mesh_y != NULL))
	{
		i = 0;
		k = 0;
		while (i < num_points)
		{
			j = 0;
			while (j < WIDTH)
			{
				mesh_y[i] = lin_space[j];
				mesh_x[i] = lin_space[k];
				i++; j++;
			}
			k++;
		}
	}
	else return 0;

	//Create the function z = f(x,y) = x^2-y^2
	float* z = (float*)malloc(num_points * sizeof(float));
	for (i = 0; i < num_points; i++) z[i] = pow(mesh_x[i], 2) - pow(mesh_y[i], 2);

	//Create data point cloud matrix
	size_t bytesD = (size_t)d_points * (size_t)3 * sizeof(float);
	float* h_D = (float*)malloc(bytesD);

	k = 0;
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			if (j == 0) h_D[k] = mesh_x[i];
			if (j == 1) h_D[k] = mesh_y[i];
			if (j == 2) h_D[k] = z[i];
			k++;
		}
	}

	//printf("Data point cloud\n");
	//printScloud(h_D, num_points, num_points);

	//Translation values
	ti[0] = 0.8f;//x
	ti[1] = -0.3f;//y
	ti[2] = 0.2f;//z

	//Rotation values (rad)
	ri[0] = 0.2f;//axis x
	ri[1] = -0.2f;//axis y
	ri[2] = 0.05f;//axis z

	float h_r[9] = {};
	float cx = (float)cos(ri[0]); float cy = (float)cos(ri[1]); float cz = (float)cos(ri[2]);
	float sx = (float)sin(ri[0]); float sy = (float)sin(ri[1]); float sz = (float)sin(ri[2]);
	h_r[0] = cy * cz; h_r[1] = (cz * sx * sy) + (cx * sz); h_r[2] = -(cx * cz * sy) + (sx * sz);
	h_r[3] = -cy * sz; h_r[4] = (cx * cz) - (sx * sy * sz); h_r[5] = (cx * sy * sz) + (cz * sx);
	h_r[6] = sy; h_r[7] = -cy * sx; h_r[8] = cx * cy;
	//printf("Ri:\n");
	//printScloud(h_r,3,3);

	//Create model point cloud matrix (target point cloud)
	//every matrix is defined using the colum-major order
	size_t bytesM = (size_t)m_points * (size_t)3 * sizeof(float);
	float* h_M = (float*)malloc(bytesM);

	//h_M = h_r*h_D
	SmatrixMul(h_r, h_D, h_M, 3, num_points, 3);
	//h_M = h_M + t
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			h_M[j + i * 3] += ti[j];
		}
	}
	//printf("\nModel point cloud\n");
	//printScloud(h_M, m_points, m_points);

	free(mesh_x); free(mesh_y); free(z);
	/////////////////////////////////////////////End of 1st//////////////////////////////////////

	//since this lines 
	//p assumes the values of D
	//q assumes the values of M
	//number of p and q points
	int p_points = d_points;
	int q_points = m_points;
	float* d_p, * d_q;
	hipMalloc(&d_p, bytesD);//p point cloud
	hipMalloc(&d_q, bytesM);//q point cloud
	//transfer data from D and M to p and q
	hipMemcpy(d_p, h_D, bytesD, hipMemcpyHostToDevice);//copy data cloud to p
	hipMemcpy(d_q, h_M, bytesM, hipMemcpyHostToDevice);//copy model cloud to q
	hipError_t err = hipSuccess;//for checking errors in kernels
	//for measuring time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float miliseconds = 0.0f;
	//cuBLAS handle
	hipblasHandle_t cublasH;
	hipblasCreate(&cublasH);
	hipblasStatus_t cublas_error;
	//cuSolver handle
	hipsolverHandle_t cusolverH;
	hipsolverDnCreate(&cusolverH);
	hipsolverStatus_t cusolver_error;

	//////////////////////////////////////////2nd:ICP Algorithm//////////////////////////////////

	//Index vector (used for correspondence)
	int* h_idx = (int*)malloc(p_points * sizeof(int));
	int* d_idx = NULL;
	hipMalloc(&d_idx, p_points * sizeof(int));

	//Q index cloud
	float* h_q_idx = (float*)malloc(bytesD);
	float* d_q_idx = NULL;
	hipMalloc(&d_q_idx, bytesD);

	//Centroids and deviations
	float* h_barP = (float*)malloc(3 * sizeof(float));
	float* h_barQ = (float*)malloc(3 * sizeof(float));
	float* h_devP = (float*)malloc(bytesD);
	float* h_devQ = (float*)malloc(bytesD);
	float* d_barP = NULL, * d_barQ = NULL;
	float* d_devP = NULL, * d_devQ = NULL;
	hipMalloc(&d_barP, 3 * sizeof(float));
	hipMalloc(&d_barQ, 3 * sizeof(float));
	hipMalloc(&d_devP, bytesD);
	hipMalloc(&d_devQ, bytesD);
	float* h_unit = (float*)malloc(p_points * sizeof(float));
	if (h_unit != NULL) for (i = 0; i < p_points; i++) h_unit[i] = 1.0f;
	float* d_unit = NULL;
	hipMalloc(&d_unit, p_points * sizeof(float));
	hipMemcpy(d_unit, h_unit, p_points * sizeof(float), hipMemcpyHostToDevice);

	//SVD
	float* h_W = (float*)malloc(sizeof(float) * 9);
	float* d_W = NULL;//3x3 matrix used in SVD
	float* d_S = NULL, * d_U = NULL, * d_VT = NULL;
	float* d_work = NULL, * d_rwork = NULL;
	int* devInfo = NULL;
	int lwork = 0;
	hipMalloc(&d_S, sizeof(float) * 3);
	hipMalloc(&d_U, sizeof(float) * 9);
	hipMalloc(&d_VT, sizeof(float) * 9);
	hipMalloc(&devInfo, sizeof(int));
	hipMalloc(&d_W, sizeof(float) * 9);
	hipsolverDnDgesvd_bufferSize(cusolverH, 3, 3, &lwork);
	hipMalloc((void**)&d_work, sizeof(float) * lwork);

	float alpha = 0, beta = 0;//for cublas routines

	//Rotation matrix and translation vector
	float* h_temp_r = (float*)malloc(9 * sizeof(float));
	float* h_temp_T = (float*)malloc(3 * sizeof(float));
	float* d_temp_r = NULL, * d_temp_T = NULL;
	hipMalloc(&d_temp_r, sizeof(float) * 9);
	hipMalloc(&d_temp_T, sizeof(float) * 3);

	//Error estimation
	float* h_error = (float*)malloc((MAX_ITER + 1) * sizeof(float));
	if (h_error != NULL) for (i = 0; i < (MAX_ITER + 1); i++) h_error[i] = 0;
	/*float* d_error = NULL;
	hipMalloc(&d_error, (MAX_ITER + 1) * sizeof(float));
	hipMemset(d_error, 0, (MAX_ITER + 1) * sizeof(float));*/
	float* d_aux = NULL;
	hipMalloc(&d_aux, bytesD);
	float partial_error = 0;

	//printf("Here starts the main loop!\n");
	int GridSize = 8;
	int BlockSize = NUM_POINTS / GridSize;
	printf("Grid Size: %d, Block Size: %d\n", GridSize, BlockSize);

	//MAIN LOOP
	int iteration = 0;
	hipEventRecord(start);
	while (iteration < MAX_ITER)
	{
		/////////////////Matching step/////////////////
		Matching << < GridSize, BlockSize >> > (d_p, d_q, q_points, d_idx);
		err = hipGetLastError();
		if (err != hipSuccess) printf("Error in matching kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();
		/*hipMemcpy(h_idx, d_idx, p_points * sizeof(int), hipMemcpyDeviceToHost);
		printf("Index  values[%d]:\n", iteration + 1);
		printIarray(h_idx, p_points);*/
		/////////////////end of matching/////////////////

		//get the Q indexed cloud
		Q_index << < GridSize, BlockSize >> > (d_q, d_idx, d_q_idx);
		err = hipGetLastError();
		if (err != hipSuccess) printf("Error in Q index kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();

		/////////////////Minimization step/////////////////

		//get the centroids
		alpha = 1 / (float)p_points; beta = 0;
		//P centroid
		cublas_error = hipblasSgemv(cublasH, HIPBLAS_OP_N, 3, p_points, &alpha, d_p, 3,
			d_unit, 1, &beta, d_barP, 1);
		if (cublas_error != HIPBLAS_STATUS_SUCCESS)
		{
			printf("Error cublas operation - P centroid\n");
			return (-1);
		}

		//Q centroid
		cublas_error = hipblasSgemv(cublasH, HIPBLAS_OP_N, 3, p_points, &alpha, d_q_idx, 3,
			d_unit, 1, &beta, d_barQ, 1);
		if (cublas_error != HIPBLAS_STATUS_SUCCESS)
		{
			printf("Error cublas operation - Q centroid\n");
			return (-1);
		}
		/*hipblasGetVector(3, sizeof(float), d_barP, 1, h_barP, 1);
		hipblasGetVector(3, sizeof(float), d_barQ, 1, h_barQ, 1);
		printf("P bar[%d]:\n", iteration + 1);
		printSarray(h_barP, 3);
		printf("Q bar[%d]:\n", iteration + 1);
		printSarray(h_barQ, 3);*/

		//get the deviation
		deviation << <GridSize, BlockSize >> > (d_p, d_q_idx, d_barP, d_barQ, d_devP, d_devQ);
		err = hipGetLastError();
		if (err != hipSuccess) printf("Error in deviation kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();

		/*hipMemcpy(h_devP, d_devP, bytesD, hipMemcpyDeviceToHost);
		printf("P dev[%d]:\n", iteration + 1);
		printScloud(h_devP, p_points, p_points);
		hipMemcpy(h_devQ, d_devQ, bytesD, hipMemcpyDeviceToHost);
		printf("Q dev[%d]:\n", iteration + 1);
		printScloud(h_devQ, p_points, p_points);*/

		//d_W = d_devM * d_devD(t)
		alpha = 1; beta = 0;
		cublas_error = hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, 3, 3, p_points,
									&alpha, d_devQ, 3, d_devP, 3, &beta, d_W, 3);
		if (cublas_error != HIPBLAS_STATUS_SUCCESS)
		{
			printf("Error cublas operation - W\n");
			return (-1);
		}

		/*hipMemcpy(h_W, d_W, 9 * sizeof(float), hipMemcpyDeviceToHost);
		printf("W[%d]:\n", iteration + 1);
		printScloud(h_W, 3, 3);*/

		//SVD W = U * S * VT
		cusolver_error = hipsolverDnSgesvd(cusolverH, 'A', 'A', 3, 3, d_W, 3, d_S, d_U, 3, d_VT, 3,
			d_work, lwork, d_rwork, devInfo);
		if (cusolver_error != HIPSOLVER_STATUS_SUCCESS)
		{
			printf("Error in SVD\n");
			return (-1);
		}

		//Calculate the temporary rotation matrix (d_temp_r)
		//R = U * VT
		alpha = 1; beta = 0;
		cublas_error = hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, 3, 3,
			&alpha, d_U, 3, d_VT, 3, &beta, d_temp_r, 3);
		if (cublas_error != HIPBLAS_STATUS_SUCCESS)
		{
			printf("Error cublas operation - Rotation Matrix\n");
			return (-1);
		}

		//T = uM - R*uD
		alpha = -1; beta = 1;
		cublas_error = hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, 1, 3,
			&alpha, d_temp_r, 3, d_barP, 3, &beta, d_barQ, 3);
		if (cublas_error != HIPBLAS_STATUS_SUCCESS)
		{
			printf("Error cublas operation - Translation Vector\n");
			return (-1);
		}
		hipblasScopy(cublasH, 3, d_barQ, 1, d_temp_T, 1);//copy the result to d_temp_T
		/////////////////end of minimization/////////////////

		/////////////////Transformation step/////////////////

		//D = R * D + T
		RyT << <GridSize, BlockSize >> > (d_temp_r, d_temp_T, d_p, d_aux);
		err = hipGetLastError();
		if (err != hipSuccess) printf("Error in RyT kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();
		hipblasScopy(cublasH, 3 * p_points, d_aux, 1, d_p, 1);

		/////////////////end of transformation step/////////////////

		/////////////////Error estimation step/////////////////
		alpha = -1;
		hipblasScopy(cublasH, 3 * p_points, d_p, 1, d_aux, 1);
		hipblasSaxpy(cublasH, 3 * p_points, &alpha, d_q_idx, 1, d_aux, 1);
		hipblasSnrm2(cublasH, 3 * p_points, d_aux, 1, &partial_error);
		h_error[iteration + 1] = partial_error / (float)sqrt(p_points);
		//printf("Current error (%d): %.4f\n", iteration + 1, h_error[iteration + 1]);
		/////////////////end of error estimation step/////////////////

		if ((h_error[iteration + 1] < 0.000001) ||
			((float)fabs((double)h_error[iteration + 1] - (double)h_error[iteration]) < 0.000001)) break;
		iteration++;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&miliseconds, start, stop);

	printf("Error:\n");
	printSarray(h_error, iteration + 1);

	printf("ICP converged successfully!\n\n");

	printf("Elapsed time: %f ms\n", miliseconds);

	//destroy handles
	hipblasDestroy(cublasH);
	hipsolverDnDestroy(cusolverH);

	//Free memory
	free(h_D), free(h_M);
	hipFree(d_p), hipFree(d_q);

	free(h_idx), hipFree(d_idx);

	free(h_q_idx), hipFree(d_q_idx);

	free(h_barP), free(h_barQ), free(h_devP), free(h_devQ), free(h_unit);
	hipFree(d_barP), hipFree(d_barQ), hipFree(d_devP), hipFree(d_devQ), hipFree(d_unit);

	free(h_W);
	hipFree(d_W), hipFree(d_S), hipFree(d_U), hipFree(d_VT);
	hipFree(d_work), hipFree(d_rwork), hipFree(devInfo);

	free(h_temp_r), free(h_temp_T);
	hipFree(d_temp_r), hipFree(d_temp_T);

	free(h_error), hipFree(d_aux);

	return 0;
}

//double matrix multiplication colum-major order
void SmatrixMul(float* A, float* B, float* C, int m, int n, int k)
{
	int i, j, q;
	float temp = 0.0f;
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < m; j++)
		{
			temp = 0.0f;
			for (q = 0; q < k; q++) temp += A[j + q * m] * B[q + i * k];
			C[j + i * m] = temp;
		}
	}
}

//print matrix
void printScloud(float* cloud, int num_points, int points2show)
{
	int i, j, offset;
	printf("x\ty\tz\n");
	if (points2show <= num_points)
	{
		for (i = 0; i < points2show; i++)
		{
			printf("%d: ", i + 1);
			for (j = 0; j < 3; j++)
			{
				offset = j + i * 3;
				printf("%.4f\t", cloud[offset]);
				if (j % 3 == 2) printf("\n");
			}
		}
	}
	else printf("The cloud can't be printed\n\n");
}

//print vector with double values
void printSarray(float* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%d: %.4f\n", i + 1, array[i]);
	}
	printf("\n");
}

//print vector with integer values
void printIarray(int* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%d: %d\n", i + 1, array[i]);
	}
	printf("\n");
}
