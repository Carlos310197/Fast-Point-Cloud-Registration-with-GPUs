#include "hip/hip_runtime.h"
//For debugging:
//nvcc ICP_standard.cu -lcublas -lcurand -lcusolver -o ICP_cuda

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>
#include <hiprand.h>
#include <hipsolver.h>
//#include <hip/device_functions.h>

//constants
#define WIDTH 32
#define NUM_POINTS WIDTH*WIDTH //width of grid (1024 points)
#define XY_min -2.0
#define XY_max 2.0
#define MAX_ITER 2

void SmatrixMul(float* A, float* B, float* C, int m, int n, int k);
void printScloud(float* cloud, int num_points, int points2show);
void printSarray(float* array, int points2show);
void printIarray(int* array, int points2show);

__global__
void Matching(float* Dt, float* M, int m, int* idx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < NUM_POINTS)
	{
		float min = 100000;
		float d;
		for (int j = 0; j < m; j++)
		{
			d = (float)sqrt((Dt[0 + i * 3] - M[0 + j * 3]) * (Dt[0 + i * 3] - M[0 + j * 3]) +
				(Dt[1 + i * 3] - M[1 + j * 3]) * (Dt[1 + i * 3] - M[1 + j * 3]) +
				(Dt[2 + i * 3] - M[2 + j * 3]) * (Dt[2 + i * 3] - M[2 + j * 3]));
			if (d < min)
			{
				min = d;
				idx[i] = j;
			}
		}
	}
}

__global__
void centroid(int n, float* D, float* M, int* idx, float* barD, float* barM, float* devD, float* devM)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	//copy D cloud to devD
	devD[0 + i * 3] = D[0 + i * 3];
	devD[1 + i * 3] = D[1 + i * 3];
	devD[2 + i * 3] = D[2 + i * 3];

	//copy M cloud to devM using the correspondence(idx)
	devM[0 + i * 3] = M[0 + idx[i] * 3];
	devM[1 + i * 3] = M[1 + idx[i] * 3];
	devM[2 + i * 3] = M[2 + idx[i] * 3];
	__syncthreads();

	for (int s = 1; s < n; s *= 2)//parallel reduction
	{
		if (i % (2 * s) == 0)
		{
			//D
			devD[0 + i * 3] += devD[0 + (i + s) * 3];
			devD[1 + i * 3] += devD[1 + (i + s) * 3];
			devD[2 + i * 3] += devD[2 + (i + s) * 3];

			//M
			devM[0 + i * 3] += devM[0 + (i + s) * 3];
			devM[1 + i * 3] += devM[1 + (i + s) * 3];
			devM[2 + i * 3] += devM[2 + (i + s) * 3];
		}
		__syncthreads();
	}

	if (i < 3)
	{
		barD[i] = devD[i] / (float)n;
		barM[i] = devM[i] / (float)n;
	}
}

__global__
void deviation(float* D, float* M, int* idx, float* barD, float* barM, float* devD, float* devM)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	devD[0 + i * 3] = D[0 + i * 3] - barD[0];
	devD[1 + i * 3] = D[1 + i * 3] - barD[1];
	devD[2 + i * 3] = D[2 + i * 3] - barD[2];

	devM[0 + i * 3] = M[0 + idx[i] * 3] - barM[0];
	devM[1 + i * 3] = M[1 + idx[i] * 3] - barM[1];
	devM[2 + i * 3] = M[2 + idx[i] * 3] - barM[2];
}

__global__
void RyT(float* R, float* T, float* P, float* Q)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	Q[0 + i * 3] = R[0 + 0 * 3] * P[0 + i * 3] + R[0 + 1 * 3] * P[1 + i * 3] + R[0 + 2 * 3] * P[2 + i * 3] + T[0];
	Q[1 + i * 3] = R[1 + 0 * 3] * P[0 + i * 3] + R[1 + 1 * 3] * P[1 + i * 3] + R[1 + 2 * 3] * P[2 + i * 3] + T[1];
	Q[2 + i * 3] = R[2 + 0 * 3] * P[0 + i * 3] + R[2 + 1 * 3] * P[1 + i * 3] + R[2 + 2 * 3] * P[2 + i * 3] + T[2];
}

__global__
void Error(int n, float* aux, float* D, float* M, int* idx, float* error, int iteration)
{
	extern __shared__ float sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	aux[0 + i * 3] = (M[0 + idx[i] * 3] - D[0 + i * 3]) * (M[0 + idx[i] * 3] - D[0 + i * 3]);
	aux[1 + i * 3] = (M[1 + idx[i] * 3] - D[1 + i * 3]) * (M[1 + idx[i] * 3] - D[1 + i * 3]);
	aux[2 + i * 3] = (M[2 + idx[i] * 3] - D[2 + i * 3]) * (M[2 + idx[i] * 3] - D[2 + i * 3]);
	__syncthreads();
	//printf("aux[%d]: %.3f %.3f %.3f\n", i, aux[0 + i * 3], aux[1 + i * 3], aux[2 + i * 3]);

	sdata[tid] = aux[0 + i * 3] + aux[1 + i * 3] + aux[2 + i * 3];
	__syncthreads();

	for (unsigned int s = 1; s < blockDim.x; s *= 2)//parallel reduction
	{
		if (tid % (2 * s) == 0)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) aux[blockIdx.x] = (float)sqrt(sdata[0] / (float)n);
	__syncthreads();

	if (i == 0)
	{
		for (unsigned int j = 0; j < gridDim.x; j++)
			error[iteration] += aux[j];
	}
}

int main()
{
	int i, j, k;
	float ti[3], ri[3];
	float lin_space[WIDTH], lenght;
	float* mesh_x = NULL, * mesh_y = NULL, * z = NULL;
	float* h_D = NULL, * h_M = NULL;

	lenght = XY_max - XY_min;

	size_t bytesD = NUM_POINTS * 3 * sizeof(float);
	size_t bytesM = NUM_POINTS * 3 * sizeof(float);

	//create an array with all points equally separated
	int n = WIDTH;
	for (i = 0; i < WIDTH; i++)
	{
		lin_space[i] = (float)XY_min + float(i) * (lenght) / (float(n) - 1.0f);
	}

	//create meshgrid for x and y coordinates
	mesh_x = (float*)malloc(NUM_POINTS * sizeof(float));
	mesh_y = (float*)malloc(NUM_POINTS * sizeof(float));
	i = 0;
	k = 0;
	while (i < NUM_POINTS)
	{
		j = 0;
		while (j < WIDTH)
		{
			mesh_y[i] = lin_space[j];
			mesh_x[i] = lin_space[k];
			i++; j++;
		}
		k++;
	}

	//Create the function z = f(x,y) = x^2-y^2
	z = (float*)malloc(NUM_POINTS * sizeof(float));
	for (i = 0; i < NUM_POINTS; i++) z[i] = (mesh_x[i])* (mesh_x[i]) - (mesh_y[i])* (mesh_y[i]);

	//Create data point cloud matrix (source point cloud)
	//points are stored in this order: x1y1z1, x2y2z2, x3y3z3, ....
	h_D = (float*)malloc(bytesD);
	k = 0;
	for (i = 0; i < NUM_POINTS; i++)
	{
		for (j = 0; j < 3; j++)
		{
			if (j == 0) h_D[k] = mesh_x[i];
			if (j == 1) h_D[k] = mesh_y[i];
			if (j == 2) h_D[k] = z[i];
			k++;
		}
	}

	/*printf("Data point cloud\n");
	print_cloud(h_D, num_points, num_points);*/

	//Create model point cloud matrix (target point cloud)
	//every matrix is defined using the colum-major order
	h_M = (float*)malloc(bytesM);

	//Translation values
	ti[0] = 1.0f;//x
	ti[1] = -0.3f;//y
	ti[2] = 0.2f;//z

	//Rotation values (rad)
	ri[0] = 1.0f;//axis x
	ri[1] = -0.5f;//axis y
	ri[2] = 0.05f;//axis z

	float h_r[9] = {};
	float cx = (float)cos(ri[0]); float cy = (float)cos(ri[1]); float cz = (float)cos(ri[2]);
	float sx = (float)sin(ri[0]); float sy = (float)sin(ri[1]); float sz = (float)sin(ri[2]);
	h_r[0] = cy * cz; h_r[1] = (cz * sx * sy) + (cx * sz); h_r[2] = -(cx * cz * sy) + (sx * sz);
	h_r[3] = -cy * sz; h_r[4] = (cx * cz) - (sx * sy * sz); h_r[5] = (cx * sy * sz) + (cz * sx);
	h_r[6] = sy; h_r[7] = -cy * sx; h_r[8] = cx * cy;
	//printf("Ri:\n");
	//printScloud(h_r, 3, 3);

	//h_M = h_r*h_D
	SmatrixMul(h_r, h_D, h_M, 3, NUM_POINTS, 3);
	//h_M = h_M + t
	for (i = 0; i < NUM_POINTS; i++)
	{
		for (j = 0; j < 3; j++)
		{
			h_M[j + i * 3] += ti[j];
		}
	}

	//printf("\nModel point cloud\n");
	//printScloud(h_M, NUM_POINTS, NUM_POINTS);

	free(mesh_x); free(mesh_y); free(z);
	/////////////////////////////////////////////End of 1st//////////////////////////////////////

	//////////////////////////////////////////2nd:ICP Algorithm//////////////////////////////////

	int D_size = NUM_POINTS;//data cloud number of points
	int M_size = NUM_POINTS;//model cloud number of points

	float* d_M = NULL;//model cloud (device)
	float* d_Dt = NULL;//data cloud transformed (device)

	int* d_idx = NULL;//index vector (used for correspondence)
	int* h_idx = NULL;//index vector (used for correspondence)

	float* d_barD = NULL, * d_barM = NULL;//centroids (device)
	float* d_devD = NULL, * d_devM = NULL;//deviations (device)

	float* d_W = NULL;//3x3 matrix used in SVD
	float* d_S = NULL;
	float* d_U = NULL;
	float* d_VT = NULL;
	float* d_work = NULL;
	float* d_rwork = NULL;
	int* devInfo = NULL;
	int lwork = 0;
	float alpha, beta;

	float* d_temp_r = NULL;
	float* d_temp_T = NULL;
	float* rep_T = NULL;

	float* d_error = NULL;
	float* h_error = NULL;

	h_idx = (int*)malloc(D_size * sizeof(int));
	hipMalloc(&d_idx, D_size * sizeof(int));

	//cuBLAS handle
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipMalloc(&d_Dt, bytesD);
	hipMalloc(&d_M, bytesM);
	hipMemcpy(d_Dt, h_D, bytesD, hipMemcpyHostToDevice);//copy data cloud to d_Dt
	hipMemcpy(d_M, h_M, bytesM, hipMemcpyHostToDevice);//copy model cloud to d_M

	hipMalloc(&d_barD, 3 * sizeof(float));
	hipMalloc(&d_barM, 3 * sizeof(float));
	hipMemset(d_barD, 0, 3 * sizeof(float));
	hipMemset(d_barM, 0, 3 * sizeof(float));

	hipMalloc(&d_devD, bytesD);
	hipMalloc(&d_devM, bytesD);
	hipMemset(d_devD, 0, bytesD);
	hipMemset(d_devM, 0, bytesD);

	//float *h_Dt = (float*)malloc(3*n*sizeof(float));

	float* h_barD, * h_barM;
	float* h_devD, * h_devM;
	h_barD = (float*)malloc(3 * sizeof(float));
	h_barM = (float*)malloc(3 * sizeof(float));
	h_devD = (float*)malloc(bytesD);
	h_devM = (float*)malloc(bytesD);

	hipMalloc(&d_S, sizeof(float) * 3);
	hipMalloc(&d_U, sizeof(float) * 9);
	hipMalloc(&d_VT, sizeof(float) * 9);
	hipMalloc(&devInfo, sizeof(int));
	hipMalloc(&d_W, sizeof(float) * 9);
	hipsolverHandle_t cusolverH;//cuSolver handle
	hipsolverDnCreate(&cusolverH);
	hipsolverDnDgesvd_bufferSize(cusolverH, 3, 3, &lwork);
	hipMalloc((void**)&d_work, sizeof(float) * lwork);

	float* h_W = (float*)malloc(sizeof(float) * 9);

	hipMalloc(&d_temp_r, sizeof(float) * 9);
	hipMalloc(&d_temp_T, sizeof(float) * 3);
	hipMalloc(&rep_T, sizeof(float) * 3 * D_size);

	hipMalloc(&d_error, MAX_ITER * sizeof(float));
	hipMemset(d_error, 0, MAX_ITER * sizeof(float));
	h_error = (float*)malloc(MAX_ITER * sizeof(float));

	hipError_t err;//for checking errors in kernels

	//printf("Here starts the main loop!\n");
	int GridSize = 8;
	int BlockSize = NUM_POINTS / GridSize;
	printf("Grid Size: %d, Block Size: %d\n", GridSize, BlockSize);

	hipblasStatus_t cublas_error;

	//MAIN LOOP
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int iteration = 0;
	hipEventRecord(start);

	while (iteration < MAX_ITER)
	{
		/////////////////matching step/////////////////
		Matching <<< GridSize, BlockSize >>> (d_Dt, d_M, M_size, d_idx);
		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error in matching kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();
		/*hipMemcpy(h_idx, d_idx, D_size*sizeof(int), hipMemcpyDeviceToHost);
		printf("Index values:\n");
		printIarray(h_idx, D_size);*/
		/////////////////end of matching/////////////////

		/////////////////minimization step/////////////////
		//hipMemcpy(d_devD, d_Dt, bytesD, hipMemcpyDeviceToDevice);
		centroid << <GridSize, BlockSize >> > (D_size, d_Dt, d_M, d_idx, d_barD, d_barM, d_devD, d_devM);
		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error in centroid kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();

		deviation << <GridSize, BlockSize >> > (d_Dt, d_M, d_idx, d_barD, d_barM, d_devD, d_devM);
		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error in deviation kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();

		hipMemcpy(h_D, d_devD, bytesD, hipMemcpyDeviceToHost);
		printf("D dev[%d]:\n", iteration + 1);
		printScloud(h_D, NUM_POINTS, NUM_POINTS);
		hipMemcpy(h_D, d_devM, bytesD, hipMemcpyDeviceToHost);
		printf("M dev[%d]:\n", iteration + 1);
		printScloud(h_D, NUM_POINTS, NUM_POINTS);

		hipMemcpy(h_barD, d_barD, 3 * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(h_barM, d_barM, 3 * sizeof(float), hipMemcpyDeviceToHost);
		printf("D centroid[%d]:\n", iteration + 1);
		printSarray(h_barD, 3);
		printf("M centroid[%d]:\n", iteration + 1);
		printSarray(h_barM, 3);

		//d_W = d_devM * d_devD(t)
		alpha = 1; beta = 0;
		cublas_error = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 3, 3, D_size,
			&alpha, d_devM, 3, d_devD, 3, &beta, d_W, 3);
		if (cublas_error != HIPBLAS_STATUS_SUCCESS)
			printf("Error in W:  %s\n", hipGetErrorString(err));

		/*hipMemcpy(h_W, d_W, 9 * sizeof(float), hipMemcpyDeviceToHost);
		printf("W:\n");
		printScloud(h_W, 3, 3);*/

		//SVD
		//d_W = d_U * d_S * d_VT
		hipsolverDnSgesvd(cusolverH, 'A', 'A', 3, 3, d_W, 3, d_S, d_U, 3, d_VT, 3,
			d_work, lwork, d_rwork, devInfo);

		//Calculate the temporary rotation matrix (d_temp_r)
		//R = U*Vt
		alpha = 1; beta = 0;
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, 3, 3,
			&alpha, d_U, 3, d_VT, 3, &beta, d_temp_r, 3);

		//T = uM - R*uD
		alpha = -1; beta = 1;
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, 1, 3,
			&alpha, d_temp_r, 3, d_barD, 3, &beta, d_barM, 3);
		hipblasScopy(handle, 3, d_barM, 1, d_temp_T, 1);//copy the result to d_temp_T
		/////////////////end of minimization/////////////////

		/////////////////transformation step/////////////////

		//D = R * D + T
		/*repmat << <GridSize, BlockSize >> > (d_temp_T, rep_T);
		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error in repmat kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();
		alpha = 1; beta = 1;
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, D_size, 3,a
			&alpha, d_temp_r, 3, d_Dt, 3, &beta, rep_T, 3);
		hipblasScopy(handle, 3 * D_size, rep_T, 1, d_Dt, 1);*/
		RyT << <GridSize, BlockSize >> > (d_temp_r, d_temp_T, d_Dt, d_devD);
		hipblasScopy(handle, 3 * D_size, d_devD, 1, d_Dt, 1);

		/////////////////end of transformation step/////////////////

		//Error
		hipMemset(d_devD, 0, bytesD);
		Error << <GridSize, BlockSize, BlockSize * sizeof(float) >> > (D_size, d_devD, d_Dt, d_M, d_idx, d_error, iteration);
		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error in error kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();

		iteration++;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipblasDestroy(handle);

	hipMemcpy(h_error, d_error, MAX_ITER * sizeof(float), hipMemcpyDeviceToHost);
	printf("Error:\n");
	printSarray(h_error, MAX_ITER);

	printf("Elapsed time: %f ms\n", milliseconds);

	free(h_D), free(h_M), free(h_idx), free(h_error), free(h_barD), free(h_barM), free(h_devD), free(h_devM);
	hipFree(d_Dt), hipFree(d_M), hipFree(d_idx), hipFree(d_barM), hipFree(d_barD), hipFree(d_devD), hipFree(d_devM);
	hipFree(d_W), hipFree(d_S), hipFree(d_U), hipFree(d_VT), hipFree(d_work), hipFree(d_rwork), hipFree(devInfo);
	hipFree(d_temp_r), hipFree(d_temp_T), hipFree(rep_T), hipFree(d_error);

	return 0;
}

//double matrix multiplication colum-major order
void SmatrixMul(float* A, float* B, float* C, int m, int n, int k)
{
	int i, j, q;
	float temp = 0.0f;
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < m; j++)
		{
			temp = 0.0f;
			for (q = 0; q < k; q++) temp += A[j + q * m] * B[q + i * k];
			C[j + i * m] = temp;
		}
	}
}

//print matrix
void printScloud(float* cloud, int num_points, int points2show)
{
	int i, j, offset;
	printf("x\ty\tz\n");
	if (points2show <= num_points)
	{
		for (i = 0; i < points2show; i++)
		{
			printf("%d: ", i + 1);
			for (j = 0; j < 3; j++)
			{
				offset = j + i * 3;
				printf("%.4f\t", cloud[offset]);
				if (j % 3 == 2) printf("\n");
			}
		}
	}
	else printf("The cloud can't be printed\n\n");
}

//print vector with double values
void printSarray(float* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%.4f ", array[i]);
	}
	printf("\n");
}

//print vector with integer values
void printIarray(int* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%d ", array[i]);
	}
	printf("\n");
}
