#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>


#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <hipsolver.h>
#include <hip/device_functions.h>

//constants
#define WIDTH 10
#define NUM_POINTS WIDTH*WIDTH //width of grid
#define XY_min -2.0
#define XY_max 2.0
#define MAX_ITER 30

void dmatrixMul(double* A, double* B, double* C, int m, int n, int k);
void print_cloud(double* cloud, int num_points, int points2show);
void print_darray(double* array, int points2show);
void print_iarray(int* array, int points2show);

__global__
void Matching(double* Dt, double* M, int m, int* d_idx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	double min = 100000;
	double d;
	for (int j = 0; j < m; j++)
	{
		d = sqrt(pow((Dt[0 + i * 3] - M[0 + j * 3]), 2) + pow((Dt[1 + i * 3] - M[1 + j * 3]), 2) + pow((Dt[2 + i * 3] - M[2 + j * 3]), 2));
		if (d < min)
		{
			min = d;
			d_idx[i] = j;
		}
	}
}

__global__
void centr_dev(double* D, double* M, int* idx, double* barD, double* barM, double* devD, double* devM)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int size = blockDim.x * gridDim.x;
	
	devM[0 + i * 3] = M[0 + idx[i] * 3];
	devM[1 + i * 3] = M[1 + idx[i] * 3];
	devM[2 + i * 3] = M[2 + idx[i] * 3];
	__syncthreads();

	for (int s = 1; s <= size; s *= 2)//parallel reduction
	{
		if (i % (2 * s) == 0)
		{
			//D
			devD[0 + i * 3] += devD[0 + (i + s) * 3];
			devD[1 + i * 3] += devD[1 + (i + s) * 3];
			devD[2 + i * 3] += devD[2 + (i + s) * 3];

			//M
			devM[0 + i * 3] += devM[0 + (i + s) * 3];
			devM[1 + i * 3] += devM[1 + (i + s) * 3];
			devM[2 + i * 3] += devM[2 + (i + s) * 3];
		}
		__syncthreads();
	}

	if (i >= 0 && i <= 2)
	{
		barD[i] = devD[i] / size;
		barM[i] = devM[i] / size;
		printf("bar D: %.3f ", barD[i]);
	}
	__syncthreads();

	devD[0 + i * 3] = D[0 + i * 3] - barD[0];
	devD[1 + i * 3] = D[1 + i * 3] - barD[1];
	devD[2 + i * 3] = D[2 + i * 3] - barD[2];

	devM[0 + i * 3] = M[0 + idx[i] * 3] - barD[0];
	devM[1 + i * 3] = M[1 + idx[i] * 3] - barD[1];
	devM[2 + i * 3] = M[2 + idx[i] * 3] - barD[2];
}

__global__
void repmat(double* vector, double* matrix)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	matrix[0 + i * 3] = vector[0];
	matrix[1 + i * 3] = vector[1];
	matrix[2 + i * 3] = vector[2];
}

__global__
void Error(double* aux, int size, double* D, double* M, int* idx, double* error, int iteration)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	aux[i] = pow(M[idx[i]] - D[i], 2);
	for (int s = 1; s <= 3 * size; s *= 2)//parallel reduction
	{
		if (i % (2 * s) == 0)
		{
			aux[i] += aux[i + s];
		}
	}
	__syncthreads();

	if (i == 0) error[iteration] = sqrt(aux[i] / size);
}

int main()
{
	int num_points, i, j, k;
	double ti[3], ri[3];
	double lin_space[WIDTH], lenght;
	double* mesh_x = NULL, * mesh_y = NULL, * z = NULL;
	double* h_D = NULL, * h_M = NULL;

	num_points = WIDTH * WIDTH;//number of points
	lenght = XY_max - XY_min;

	size_t bytesD = num_points * 3 * sizeof(double);
	size_t bytesM = num_points * 3 * sizeof(double);

	//create an array with all points equally separated
	int n = WIDTH;
	for (i = 0; i < WIDTH; i++)
	{
		lin_space[i] = XY_min + double(i) * (lenght) / (double(n) - 1.0f);
	}

	//create meshgrid for x and y coordinates
	mesh_x = (double*)malloc(num_points * sizeof(double));
	mesh_y = (double*)malloc(num_points * sizeof(double));
	i = 0;
	k = 0;
	while (i < num_points)
	{
		j = 0;
		while (j < WIDTH)
		{
			mesh_y[i] = lin_space[j];
			mesh_x[i] = lin_space[k];
			i++; j++;
		}
		k++;
	}

	//Create the function z = f(x,y) = x^2-y^2
	z = (double*)malloc(num_points * sizeof(double));
	for (i = 0; i < num_points; i++) z[i] = pow(mesh_x[i], 2) - pow(mesh_y[i], 2);

	//Create data point cloud matrix (source point cloud)
	//points are stored in this order: x1y1z1, x2y2z2, x3y3z3, ....
	h_D = (double*)malloc(bytesD);
	k = 0;
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			if (j == 0) h_D[k] = mesh_x[i];
			if (j == 1) h_D[k] = mesh_y[i];
			if (j == 2) h_D[k] = z[i];
			k++;
		}
	}

	/*printf("Data point cloud\n");
	print_cloud(h_D, num_points, num_points);*/

	//Create model point cloud matrix (target point cloud)
	//every matrix is defined using the colum-major order
	h_M = (double*)malloc(bytesM);

	//Translation values
	ti[0] = 1.0f;//x
	ti[1] = -0.3f;//y
	ti[2] = 0.2f;//z

	//Rotation values (rad)
	ri[0] = 1.0f;//axis x
	ri[1] = -0.5f;//axis y
	ri[2] = 0.05f;//axis z

	double h_rx[9] = {
		1.0f, 0.0f, 0.0f,
		0.0f, cos(ri[0]), sin(ri[0]),
		0.0f, -sin(ri[0]), cos(ri[0]) };
	double h_ry[9] = {
		cos(ri[1]), 0.0f, -sin(ri[1]),
		0.0f, 1, 0.0f,
		sin(ri[1]), 0.0f, cos(ri[1]) };
	double h_rz[9] = {
		cos(ri[2]), sin(ri[2]),0.0f,
		-sin(ri[2]), cos(ri[2]), 0.0f,
		0.0f, 0.0f, 1.0f };

	//calculate the rotation matrix h_r
	/*double h_r[9] = {};
	dmatrixMul(h_rx, h_ry, h_r, 3, 3, 3);
	dmatrixMul(h_r, h_rz, h_r, 3, 3, 3);*/
	double h_r[9] = {0.8765,-0.3759,0.3008,-0.0439,0.5598,0.8275,-0.4794,-0.7385,0.4742};

	/*printf("Rx:\n");
	print_cloud(h_rx,3,3);
	printf("Ry:\n");
	print_cloud(h_ry,3,3);
	printf("Rz:\n");
	print_cloud(h_rz,3,3);
	printf("Rotation Matrix:\n");
	print_cloud(h_r,3,3);*/

	//h_M = h_r*h_D
	dmatrixMul(h_r, h_D, h_M, 3, num_points, 3);
	//h_M = h_M + t
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			h_M[j + i * 3] += ti[j];
		}
	}

	/*printf("\nModel point cloud\n");
	print_cloud(h_M, num_points, num_points);*/

	free(mesh_x); free(mesh_y); free(z);
	/////////////////////////////////////////////End of 1st//////////////////////////////////////

	//////////////////////////////////////////2nd:ICP Algorithm//////////////////////////////////

	int D_size = NUM_POINTS;//data cloud number of points
	int M_size = NUM_POINTS;//model cloud number of points
	
	double* d_M = NULL;//model cloud (device)
	double* d_Dt = NULL;//data cloud transformed (device)
	
	int* d_idx = NULL;//index vector (used for correspondence)
	int* h_idx = NULL;//index vector (used for correspondence)
	
	double* d_barD = NULL, * d_barM = NULL;//centroids (device)
	double* d_devD = NULL, * d_devM = NULL;//deviations (device)
	
	double* d_W = NULL;//3x3 matrix used in SVD
	double* d_S = NULL;
	double* d_U = NULL;
	double* d_VT = NULL;
	double* d_work = NULL;
	double* d_rwork = NULL;
	int* devInfo = NULL;
	int lwork = 0;
	double alpha, beta;

	double* d_temp_r = NULL;
	double* d_temp_T = NULL;
	double* rep_T = NULL;

	double* d_error = NULL;
	double* h_error = NULL;

	h_idx = (int*)malloc(D_size * sizeof(int));
	hipMalloc(&d_idx, D_size * sizeof(int));

	//cuBLAS handle
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipMalloc(&d_Dt, bytesD);
	hipMalloc(&d_M, bytesM);
	hipMemcpy(d_Dt, h_D, bytesD, hipMemcpyHostToDevice);
	hipMemcpy(d_M, h_M, bytesM, hipMemcpyHostToDevice);

	hipMalloc(&d_barD, 3 * sizeof(double));
	hipMalloc(&d_barM, 3 * sizeof(double));
	hipMemset(d_barD, 0, 3 * sizeof(double));
	hipMemset(d_barM, 0, 3 * sizeof(double));
	hipMalloc(&d_devD, bytesD);
	hipMalloc(&d_devM, bytesD);
	
	double* h_barD, * h_barM;
	double* h_devD, * h_devM;
	h_barD = (double*)malloc(3 * sizeof(double));
	h_barM = (double*)malloc(3 * sizeof(double));
	d_devD = (double*)malloc(bytesD);
	d_devM = (double*)malloc(bytesD);

	hipMalloc(&d_S, sizeof(double) * 3);
	hipMalloc(&d_U, sizeof(double) * 9);
	hipMalloc(&d_VT, sizeof(double) * 9);
	hipMalloc(&devInfo, sizeof(int));
	hipMalloc(&d_W, sizeof(double) * 9);
	hipsolverHandle_t cusolverH;//cuSolver handle
	hipsolverDnCreate(&cusolverH);
	hipsolverDnDgesvd_bufferSize(cusolverH, 3, 3, &lwork);
	hipMalloc((void**)&d_work, sizeof(double) * lwork);

	hipMalloc(&d_temp_r, sizeof(double) * 9);
	hipMalloc(&d_temp_T, sizeof(double) * 3);
	hipMalloc(&rep_T, sizeof(double) * 3 * D_size);

	hipMalloc(&d_error, MAX_ITER * sizeof(double));
	h_error = (double*)malloc(MAX_ITER * sizeof(double));

	hipError_t err;

	//printf("Here starts the main loop!\n");
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int iteration = 0;
	hipEventRecord(start);
	//MAIN LOOP
	while (1)
	{
		/////////////////matching step/////////////////
		Matching<<<1,D_size>>>(d_Dt, d_M, M_size, d_idx);
		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error in matching kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();
		/////////////////end of matching/////////////////

		/////////////////minimization step/////////////////
		hipMemcpy(d_devD, d_Dt, bytesD, hipMemcpyDeviceToDevice);
		centr_dev<<<1,D_size>>>(d_Dt, d_M, d_idx, d_barD, d_barM, d_devD, d_devM);
		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error in centroid kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();
		/*cudaMemcpy(h_D, d_devD, bytesD, cudaMemcpyDeviceToHost);
		printf("D dev:\n");
		print_cloud(h_D, D_size, 20);*/

		hipMemcpy(h_barD, d_barD, bytesD, hipMemcpyDeviceToHost);
		hipMemcpy(h_barM, d_barM, bytesD, hipMemcpyDeviceToHost);
		printf("D centroid:\n");
		print_darray(h_barD, 3);
		printf("M centroid:\n");
		print_darray(h_barM, 3);

		//d_W = d_devM * d_devD(t)
		alpha = 1; beta = 0;
		hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 3, 3, D_size,
			&alpha, d_devM, 3, d_devD, 3, &beta, d_W, 3);

		//SVD
		//d_W = d_U * d_S * d_VT
		hipsolverDnDgesvd(cusolverH, 'A', 'A', 3, 3, d_W, 3, d_S, d_U, 3, d_VT, 3,
			d_work, lwork, d_rwork, devInfo);

		//Calculate the temporary rotation matrix (d_temp_r)
		//R = U*Vt
		alpha = 1; beta = 0;
		hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, 3, 3,
			&alpha, d_U, 3, d_VT, 3, &beta, d_temp_r, 3);

		//T = uM - R*uD
		alpha = -1; beta = 1;
		hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, 1, 3,
			&alpha, d_temp_r, 3, d_barD, 3, &beta, d_barM, 3);
		hipblasDcopy(handle, 3, d_barM, 1, d_temp_T, 1);//copy the result to d_temp_T
		/////////////////end of minimization/////////////////

		/////////////////transformation step/////////////////

		//D = R * D + T
		repmat <<<1, D_size>>> (d_temp_T, rep_T);
		hipDeviceSynchronize();
		alpha = 1; beta = 1;
		hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, D_size, 3,
			&alpha, d_temp_r, 3, d_Dt, 3, &beta, rep_T, 3);
		hipblasDcopy(handle, 3 * D_size, rep_T, 1, d_Dt, 1);

		/////////////////end of transformation step/////////////////

		//Error
		Error <<<3, D_size>>> (d_devD, D_size, d_Dt, d_M, d_idx, d_error, iteration);
		hipDeviceSynchronize();

		iteration++;
		if (iteration > MAX_ITER + 1) break;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipblasDestroy(handle);

	hipMemcpy(h_error, d_error, MAX_ITER * sizeof(double), hipMemcpyDeviceToHost);
	printf("Error:\n");
	print_darray(h_error, iteration);

	printf("Elapsed time: %f ms\n", milliseconds);

	free(h_D), free(h_M), free(h_idx), free(h_error), free(h_barD), free(h_barM), free(h_devD), free(h_devM);
	hipFree(d_Dt), hipFree(d_M), hipFree(d_idx), hipFree(d_barM), hipFree(d_barD), hipFree(d_devD), hipFree(d_devM);
	hipFree(d_W), hipFree(d_S), hipFree(d_U), hipFree(d_VT), hipFree(d_work), hipFree(d_rwork), hipFree(devInfo);
	hipFree(d_temp_r), hipFree(d_temp_T), hipFree(rep_T), hipFree(d_error);

	return 0;
}

//double matrix multiplication
void dmatrixMul(double* A, double* B, double* C, int m, int n, int k)
{
	int i, j, q;
	double temp;
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < m; j++)
		{
			temp = 0.0f;
			for (q = 0; q < k; q++) temp += double(A[j + q * m] * B[q + i * k]);
			C[j + i * m] = (double)temp;
		}
	}
}

//print matrix
void print_cloud(double* cloud, int num_points, int points2show)//
{
	int i, j, offset;
	printf("x\ty\tz\n");
	if (points2show <= num_points)
	{
		for (i = 0; i < points2show; i++)
		{
			for (j = 0; j < 3; j++)
			{
				offset = j + i * 3;
				printf("%.4f\t", cloud[offset]);
				if (j % 3 == 2) printf("\n");
			}
		}
	}
	else printf("The cloud can't be printed\n\n");
}

//print vector with double values
void print_darray(double* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%.3f ", array[i]);
	}
	printf("\n");
}

//print vector with integer values
void print_iarray(int* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%d ", array[i]);
	}
	printf("\n");
}

