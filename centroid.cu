#include "hip/hip_runtime.h"
//code to test the centroid kernel
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>
#include <hiprand.h>
#include <hipsolver.h>
#define NUM_POINTS 2048

__global__
void Centroid(float* cloud, float* bar)
{
    __shared__ float sdatax[NUM_POINTS];
    __shared__ float sdatay[NUM_POINTS];
    __shared__ float sdataz[NUM_POINTS];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdatax[tid] = cloud[i * 3 + 0];
    sdatay[tid] = cloud[i * 3 + 1];
    sdataz[tid] = cloud[i * 3 + 2];
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2)
    {
        if (tid % (2 * s) == 0) 
        {
            sdatax[tid] += sdatax[tid + s];
            sdatay[tid] += sdatay[tid + s];
            sdataz[tid] += sdataz[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        bar[0 + 3 * blockIdx.x] = sdatax[tid];
        bar[1 + 3 * blockIdx.x] = sdatay[tid];
        bar[2 + 3 * blockIdx.x] = sdataz[tid];
        //printf("bar[2]: %.3f\n", bar[2 + 3 * blockIdx.x]);
    }
}

int main(void)
{
    
    float* h_cloud = NULL, * d_cloud = NULL;
    float* h_bar = NULL, * d_bar = NULL;
    float* h_bar_2 = NULL, * d_bar_2 = NULL;

    int GridSize = 8;
    int BlockSize = NUM_POINTS / GridSize;

    size_t size_cloud = 3 * NUM_POINTS * sizeof(float);
    //size_t size_bar = 3 * GridSize * sizeof(float);

    h_cloud = (float*)malloc(size_cloud);
    hipMalloc(&d_cloud, size_cloud);

    h_bar = (float*)malloc(size_cloud);
    hipMalloc(&d_bar, size_cloud);

    for (unsigned int i = 0; i < NUM_POINTS; i++)
    {
        h_cloud[0 + i * 3] = 1.0f;//all the x values get 1
        h_cloud[1 + i * 3] = 2.0f;//all the y values get 2
        h_cloud[2 + i * 3] = 3.0f;//all the z values get 3
    }

    hipMemcpy(d_cloud, h_cloud, size_cloud, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    Centroid <<< GridSize, BlockSize >>> (d_cloud, d_bar);
    Centroid << < 1, BlockSize >> > (d_bar, d_bar);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("Error in kernel: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float miliseconds1 = 0;
    hipEventElapsedTime(&miliseconds1, start, stop);

    hipMemcpy(h_bar, d_bar, 3 * sizeof(float), hipMemcpyDeviceToHost);

    printf("bar 1:\n");
    for (int i = 0; i < 3; i++) printf("%.3f ", h_bar[i]);
    printf("\n");

    //cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;
    float* d_x = NULL;
    hipMalloc(&d_x, 3 * sizeof(float));
    hipMemset(d_x, 1, 3 * sizeof(float));
    h_bar_2 = (float*)malloc(3 * sizeof(float));
    hipMalloc(&d_bar_2, 3 * sizeof(float));

    hipEventRecord(start);
    hipblasSgemv(handle, HIPBLAS_OP_N, 3, NUM_POINTS, &alpha, d_cloud, 3, d_x, 1, &beta, d_bar_2, 1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float miliseconds2 = 0;
    hipEventElapsedTime(&miliseconds2, start, stop);

    hipMemcpy(h_bar_2, d_bar_2, 3 * sizeof(float), hipMemcpyDeviceToHost);
    printf("bar 2:\n");
    for (int i = 0; i < 3; i++) printf("%.3f ", h_bar_2[i]);
    printf("\n");

    printf("\nElapsed time 1: %f ms\n", miliseconds1);
    printf("\nElapsed time 2: %f ms\n", miliseconds2);

    hipblasDestroy(handle);
    hipFree(d_cloud), hipFree(d_bar), hipFree(d_bar_2);
    free(h_cloud), free(h_bar), free(h_bar_2);
    hipFree(d_x);

    return 1;
}
