//For debugging:
//nvcc ICP_standard.cu -lcublas -lcurand -lcusolver -o ICP_cuda

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

//#include <cublas.h>
//#include <curand.h>
//#include <cusolverDn.h>
//#include <device_functions.h>

//constants
#define WIDTH 32
#define XY_min -2.0
#define XY_max 2.0
#define MAX_ITER 40

void SmatrixMul(float* A, float* B, float* C, int m, int n, int k);
void printScloud(float* cloud, int num_points, int points2show);
void printSarray(float* array, int points2show);
void printIarray(int* array, int points2show);

//idx has to allocate mxn values
//d has to allocate mxn values

__global__
void knn(float* Dt, int n, float* M, int m, int* idx, int k, float* d)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j, s;
	float key = 0.0f;

	for (j = 0; j < m; j++)
		d[j + i * m] = (float)sqrt(pow((Dt[0 + i * 3] - M[0 + j * 3]), 2) + pow((Dt[1 + i * 3] - M[1 + j * 3]), 2) + pow((Dt[2 + i * 3] - M[2 + j * 3]), 2));

	__syncthreads();

	//sort the distances saving the index values (insertion sort)
	//each thread is in charge of a distance sort
	float* arr = d + i * m;
	int* r = idx + i * m;
	r[0] = 0;
	for (s = 0; s < m; s++)
	{
		key = arr[s];
		j = s - 1;
		while (j >= 0 && arr[j] > key)
		{
			arr[j + 1] = arr[j];
			r[j + 1] = r[j];
			j--;
		}
		arr[j + 1] = key;
		r[j + 1] = s;
	}
}

int main(void)
{
	int num_points, i, j, k;
	float ti[3], ri[3];
	float lin_space[WIDTH], lenght;

	num_points = WIDTH * WIDTH;//number of points
	lenght = XY_max - XY_min;

	//for this specific case the number of points of the 2 clouds are the same
	int d_points = num_points;
	int m_points = num_points;

	////////////////1st:Creation of the synthetic data//////////////

	//create an array with all points equally separated
	int n = WIDTH;
	for (i = 0; i < WIDTH; i++)
	{
		lin_space[i] = (float)XY_min + ((float)i * (float)lenght) / (float(n) - 1.0f);
	}

	//create the meshgrid
	float* mesh_x = (float*)malloc(num_points * sizeof(float));
	float* mesh_y = (float*)malloc(num_points * sizeof(float));

	if ((mesh_x != NULL) && (mesh_y != NULL))
	{
		i = 0;
		k = 0;
		while (i < num_points)
		{
			j = 0;
			while (j < WIDTH)
			{
				mesh_y[i] = lin_space[j];
				mesh_x[i] = lin_space[k];
				i++; j++;
			}
			k++;
		}
	}
	else return 0;

	//Create the function z = f(x,y) = x^2-y^2
	float* z = (float*)malloc(num_points * sizeof(float));
	for (i = 0; i < num_points; i++) z[i] = pow(mesh_x[i], 2) - pow(mesh_y[i], 2);

	//Create data point cloud matrix
	size_t bytesD = (size_t)d_points * (size_t)3 * sizeof(float);
	float* h_D = (float*)malloc(bytesD);

	k = 0;
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			if (j == 0) h_D[k] = mesh_x[i];
			if (j == 1) h_D[k] = mesh_y[i];
			if (j == 2) h_D[k] = z[i];
			k++;
		}
	}

	//printf("Data point cloud\n");
	//printScloud(h_D, num_points, num_points);

	//Translation values
	ti[0] = 1.0f;//x
	ti[1] = -0.3f;//y
	ti[2] = 0.2f;//z

	//Rotation values (rad)
	ri[0] = 1.0f;//axis x
	ri[1] = -0.5f;//axis y
	ri[2] = 0.05f;//axis z

	float h_r[9] = {};
	float cx = (float)cos(ri[0]); float cy = (float)cos(ri[1]); float cz = (float)cos(ri[2]);
	float sx = (float)sin(ri[0]); float sy = (float)sin(ri[1]); float sz = (float)sin(ri[2]);
	h_r[0] = cy * cz; h_r[1] = (cz * sx * sy) + (cx * sz); h_r[2] = -(cx * cz * sy) + (sx * sz);
	h_r[3] = -cy * sz; h_r[4] = (cx * cz) - (sx * sy * sz); h_r[5] = (cx * sy * sz) + (cz * sx);
	h_r[6] = sy; h_r[7] = -cy * sx; h_r[8] = cx * cy;
	//printf("Ri:\n");
	//printScloud(h_r,3,3);

	//Create model point cloud matrix (target point cloud)
	//every matrix is defined using the colum-major order
	size_t bytesM = (size_t)m_points * (size_t)3 * sizeof(float);
	float* h_M = (float*)malloc(bytesM);

	//h_M = h_r*h_D
	SmatrixMul(h_r, h_D, h_M, 3, num_points, 3);
	//h_M = h_M + t
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			h_M[j + i * 3] += ti[j];
		}
	}
	//printf("\nModel point cloud\n");
	//printScloud(h_M, NUM_POINTS, NUM_POINTS);

	/////////End of 1st/////////

	int GridSize = 8;
	int BlockSize = num_points / GridSize;
	printf("Grid Size: %d, Block Size: %d\n", GridSize, BlockSize);

	//since this lines 
	//p assumes the value of D
	//q assumes the value of M
	//number of p and q points
	int p_points = d_points;
	int q_points = m_points;
	float* d_p, *d_q; 
	hipMalloc(&d_p, bytesD);//p points cloud
	hipMalloc(&d_q, bytesM);//p points cloud//q point cloud
	//transfer data from D and M to p and q
	hipMemcpy(d_p, h_D, bytesD, hipMemcpyHostToDevice);//copy data cloud to p
	hipMemcpy(d_q, h_M, bytesM, hipMemcpyHostToDevice);//copy model cloud to q
	hipError_t err;//for checking errors in kernels

	/////////2nd: Normals estimation/////////
	int* d_idx;
	float* d_dist;
	hipMalloc(&d_idx, (size_t)p_points * (size_t)q_points * sizeof(int));
	hipMalloc(&d_dist, (size_t)p_points * (size_t)q_points * sizeof(float));
	k = 4;//number of nearest neighbors
	knn <<< GridSize, BlockSize >>> (d_q, q_points, d_q, q_points, d_idx, k + 1, d_dist);
	if (err != hipSuccess)
		printf("Error in matching kernel: %s\n", hipGetErrorString(err));
	hipDeviceSynchronize();
	/////////End of 2nd/////////

	free(mesh_x), free(mesh_y), free(z);
	free(h_M), free(h_D);
	//cudaFree(d_p), cudaFree(d_q);
	//cudaFree(d_idx), cudaFree(d_dist);
	return 0;
}

//double matrix multiplication colum-major order
void SmatrixMul(float* A, float* B, float* C, int m, int n, int k)
{
	int i, j, q;
	float temp = 0.0f;
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < m; j++)
		{
			temp = 0.0f;
			for (q = 0; q < k; q++) temp += A[j + q * m] * B[q + i * k];
			C[j + i * m] = temp;
		}
	}
}

//print matrix
void printScloud(float* cloud, int num_points, int points2show)
{
	int i, j, offset;
	printf("x\ty\tz\n");
	if (points2show <= num_points)
	{
		for (i = 0; i < points2show; i++)
		{
			for (j = 0; j < 3; j++)
			{
				offset = j + i * 3;
				printf("%.4f\t", cloud[offset]);
				if (j % 3 == 2) printf("\n");
			}
		}
	}
	else printf("The cloud can't be printed\n\n");
}

//print vector with double values
void printSarray(float* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%.4f ", array[i]);
	}
	printf("\n");
}

//print vector with integer values
void printIarray(int* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%d ", array[i]);
	}
	printf("\n");
}
