#include "hip/hip_runtime.h"
//For debugging:
//nvcc ICP_standard.cu -lcublas -lcurand -lcusolver -o ICP_cuda

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>
#include <hiprand.h>
#include <hipsolver.h>
//#include <hip/device_functions.h>

//constants
#define WIDTH 32
#define XY_min -2.0
#define XY_max 2.0
#define MAX_ITER 40

void SmatrixMul(float* A, float* B, float* C, int m, int n, int k);
void printScloud(float* cloud, int num_points, int points2show);
void printSarray(float* array, int points2show);
void printIarray(int* array, int points2show);

//idx has to allocate mxn values
//d has to allocate mxn values
__global__
void knn(float* Dt, int n, float* M, int m, int* idx, int k, float* d)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j, s;
	float key = 0.0f;

	for (j = 0; j < m; j++)
		d[j + i * m] = (float)sqrt(pow((Dt[0 + i * 3] - M[0 + j * 3]), 2) + pow((Dt[1 + i * 3] - M[1 + j * 3]), 2) + pow((Dt[2 + i * 3] - M[2 + j * 3]), 2));

	__syncthreads();

	//sort the distances saving the index values (insertion sort)
	//each thread is in charge of a distance sort
	float* arr = d + i * m;
	int* r = idx + i * m;
	r[0] = 0;
	for (s = 0; s < m; s++)
	{
		key = arr[s];
		j = s - 1;
		while (j >= 0 && arr[j] > key)
		{
			arr[j + 1] = arr[j];
			r[j + 1] = r[j];
			j--;
		}
		arr[j + 1] = key;
		r[j + 1] = s;
	}
}

__global__
void Normals(float* q, int* neighbors, int n, int m, int k, float* normals)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = 0, stride = 0;
	float bar[3] = {};

	//step 1: find the centroid of the k nearest neighbors
	for (j = 1; j < k + 1; j++)
	{
		stride = neighbors[j + i * m];//neighbors are stored row-major
		bar[0] += (q[0 + stride * 3] / (float)k);//q is stored colum-major (x1y1z1 ...)
		bar[1] += (q[1 + stride * 3] / (float)k);
		bar[2] += (q[2 + stride * 3] / (float)k);
	}
	__syncthreads();

	float xi = 0.0f, yi = 0.0f, zi = 0.0f;
	float A[9] = {};

	//step 2: find the covariance matrix A
	for (j = 1; j < k + 1; j++)
	{
		stride = neighbors[j + i * m];
		xi = q[0 + stride * 3];
		yi = q[1 + stride * 3];
		zi = q[2 + stride * 3];
		//place the values of the upper triangular matrix A only
		A[0] += (xi - bar[0]) * (xi - bar[0]);
		A[1] += (xi - bar[0]) * (yi - bar[1]);
		A[2] += (xi - bar[0]) * (zi - bar[2]);
		A[4] += (yi - bar[1]) * (yi - bar[1]);
		A[5] += (yi - bar[1]) * (zi - bar[2]);
		A[8] += (zi - bar[2]) * (zi - bar[2]);
	}
	__syncthreads();

	//step 3: compute the eigenvectors of A
	float p1 = A[1] * A[1] + A[2] * A[2] + A[5] * A[5];
	float qi = 0.0f, p2 = 0.0f, p = 0.0f, r = 0.0f, phi = 0.0f;
	float eigen[3] = {};

	qi = (A[0] + A[4] + A[8]) / 3.0f;//trace(A)
	p2 = (A[0] - qi) * (A[0] - qi) +
		(A[4] - qi) * (A[4] - qi) +
		(A[8] - qi) * (A[8] - qi) + 2 * p1;
	p = (float)sqrt(p2 / 6.0f);
	r = ((float)1 / (2 * p * p * p)) *
		((A[0] - qi) * ((A[4] - qi) * (A[8] - qi) - A[5] * A[5])
			- A[1] * (A[1] * (A[8] - qi) - A[2] * A[5])
			+ A[2] * (A[1] * A[5] - A[2] * (A[4] - qi)));
	if (r <= -1) phi = (float)M_PI / 3.0f;
	else if (r >= 1) phi = 0.0f;
	else  phi = (float)acos(r) / 3.0f;

	//the eigenvalues satisfy eig3 <= eig2 <= eig1
	//eigen[0] = qi + 2 * p * (float)cos(phi);//eigenvalue 1
	eigen[2] = qi + 2 * p * (float)cos(phi + (2 * M_PI / 3));//eigenvalue 3
	//eigen[1] = 3 * qi - eigen[0] - eigen[2];//eigenvalue 2

	A[3] = A[1];
	A[0] -= eigen[2];
	A[4] -= eigen[2];
	float aux = A[3] / A[0];
	A[3] -= A[0] * aux;
	A[4] -= A[1] * aux;
	A[5] -= A[2] * aux;

	float eigenvector[3] = { 1.0f,1.0f,1.0f };
	eigenvector[1] = -A[5] / A[4];
	eigenvector[0] = -(A[1] * eigenvector[1] + A[2] * eigenvector[2]) / A[0];

	normals[0 + i * 3] = eigenvector[0];
	normals[1 + i * 3] = eigenvector[1];
	normals[2 + i * 3] = eigenvector[2];
}

__global__
void Matching(float* Dt, float* M, int m, int* idx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float min = 100000;
	float d;
	for (int j = 0; j < m; j++)
	{
		d = (float)sqrt(pow((Dt[0 + i * 3] - M[0 + j * 3]), 2) + pow((Dt[1 + i * 3] - M[1 + j * 3]), 2) + pow((Dt[2 + i * 3] - M[2 + j * 3]), 2));
		if (d < min)
		{
			min = d;
			idx[i] = j;
		}
	}
}

//C has to be stored in column-major order
__global__
void Cxb(float* p, int n, float* q, int m, int* idx, float* normals, float* cn, float* C_total, float* b_total, float* C, float* b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = idx[i];
	float* aux = (float*)malloc((size_t)n * sizeof(float));
	cn[0 + i * 6] = p[1 + i * 3] * normals[2 + stride * 3] -
					p[2 + i * 3] * normals[1 + stride * 3];//cix
	cn[1 + i * 6] = p[2 + i * 3] * normals[0 + stride * 3] -
					p[0 + i * 3] * normals[2 + stride * 3];//ciy
	cn[2 + i * 6] = p[0 + i * 3] * normals[1 + stride * 3] -
					p[1 + i * 3] * normals[0 + stride * 3];//ciz
	cn[3 + i * 6] = normals[0 + stride * 3];//nix
	cn[4 + i * 6] = normals[1 + stride * 3];//niy
	cn[5 + i * 6] = normals[2 + stride * 3];//niz

	C_total[0 + i * 6] = cn[0 + i * 6] * cn[0 + i * 6]; C_total[1 + i * 6] = cn[0 + i * 6] * cn[1 + i * 6]; C_total[2 + i * 6] = cn[0 + i * 6] * cn[2 + i * 6];
	C_total[3 + i * 6] = cn[0 + i * 6] * cn[3 + i * 6]; C_total[4 + i * 6] = cn[0 + i * 6] * cn[4 + i * 6]; C_total[5 + i * 6] = cn[0 + i * 6] * cn[5 + i * 6];
	C_total[6 + i * 6] = cn[1 + i * 6] * cn[1 + i * 6]; C_total[7 + i * 6] = cn[1 + i * 6] * cn[2 + i * 6]; C_total[8 + i * 6] = cn[1 + i * 6] * cn[3 + i * 6];
	C_total[9 + i * 6] = cn[1 + i * 6] * cn[4 + i * 6]; C_total[10 + i * 6] = cn[1 + i * 6] * cn[5 + i * 6]; C_total[11 + i * 6] = cn[2 + i * 6] * cn[2 + i * 6];
	C_total[12 + i * 6] = cn[2 + i * 6] * cn[3 + i * 6]; C_total[13 + i * 6] = cn[2 + i * 6] * cn[4 + i * 6]; C_total[14 + i * 6] = cn[2 + i * 6] * cn[5 + i * 6];
	C_total[15 + i * 6] = cn[3 + i * 6] * cn[3 + i * 6]; C_total[16 + i * 6] = cn[3 + i * 6] * cn[4 + i * 6]; C_total[17 + i * 6] = cn[3 + i * 6] * cn[5 + i * 6];
	C_total[18 + i * 6] = cn[4 + i * 6] * cn[4 + i * 6]; C_total[19 + i * 6] = cn[4 + i * 6] * cn[5 + i * 6]; C_total[20 + i * 6] = cn[5 + i * 6] * cn[5 + i * 6];

	aux[i] = (p[0 + i * 3] - q[0 + i * 3]) * cn[3 + i * 6] +
			 (p[1 + i * 3] - q[1 + i * 3]) * cn[4 + i * 6] +
			 (p[2 + i * 3] - q[2 + i * 3]) * cn[5 + i * 6];

	b_total[0 + i * 6] = cn[0 + i * 6] * aux[i]; b_total[1 + i * 6] = cn[1 + i * 6] * aux[i]; b_total[2 + i * 6] = cn[2 + i * 6] * aux[i];
	b_total[3 + i * 6] = cn[3 + i * 6] * aux[i]; b_total[4 + i * 6] = cn[4 + i * 6] * aux[i]; b_total[5 + i * 6] = cn[5 + i * 6] * aux[i];
	__syncthreads();

	for (int s = 1; s < n; s *= 2)//parallel reduction
	{
		if (i % (2 * s) == 0)
		{
			C_total[0 + i * 6] += C_total[0 + (i + s) * 6]; C_total[1 + i * 6] += C_total[1 + (i + s) * 6]; C_total[2 + i * 6] += C_total[2 + (i + s) * 6];
			C_total[3 + i * 6] += C_total[3 + (i + s) * 6]; C_total[4 + i * 6] += C_total[4 + (i + s) * 6]; C_total[5 + i * 6] += C_total[5 + (i + s) * 6];
			C_total[6 + i * 6] += C_total[6 + (i + s) * 6]; C_total[7 + i * 6] += C_total[7 + (i + s) * 6]; C_total[8 + i * 6] += C_total[8 + (i + s) * 6];
			C_total[9 + i * 6] += C_total[9 + (i + s) * 6]; C_total[10 + i * 6] += C_total[10 + (i + s) * 6]; C_total[11 + i * 6] += C_total[11 + (i + s) * 6];
			C_total[12 + i * 6] += C_total[12 + (i + s) * 6]; C_total[13 + i * 6] += C_total[13 + (i + s) * 6]; C_total[14 + i * 6] += C_total[14 + (i + s) * 6];
			C_total[15 + i * 6] += C_total[15 + (i + s) * 6]; C_total[16 + i * 6] += C_total[16 + (i + s) * 6]; C_total[17 + i * 6] += C_total[17 + (i + s) * 6];
			C_total[18 + i * 6] += C_total[18 + (i + s) * 6]; C_total[19 + i * 6] += C_total[19 + (i + s) * 6]; C_total[20 + i * 6] += C_total[20 + (i + s) * 6];

			b_total[0 + i * 6] += b_total[0 + (i + s) * 6]; b_total[1 + i * 6] += b_total[1 + (i + s) * 6]; b_total[2 + i * 6] += b_total[2 + (i + s) * 6];
			b_total[3 + i * 6] += b_total[3 + (i + s) * 6]; b_total[4 + i * 6] += b_total[4 + (i + s) * 6]; b_total[5 + i * 6] += b_total[5 + (i + s) * 6];
		}
		__syncthreads();
	}

}

__global__
void RyT(float* R, float* T, float* P, float* Q)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	Q[0 + i * 3] = R[0 + 0 * 3] * P[0 + i * 3] + R[0 + 1 * 3] * P[1 + i * 3] + R[0 + 2 * 3] * P[2 + i * 3] + T[0];
	Q[1 + i * 3] = R[1 + 0 * 3] * P[0 + i * 3] + R[1 + 1 * 3] * P[1 + i * 3] + R[1 + 2 * 3] * P[2 + i * 3] + T[1];
	Q[2 + i * 3] = R[2 + 0 * 3] * P[0 + i * 3] + R[2 + 1 * 3] * P[1 + i * 3] + R[2 + 2 * 3] * P[2 + i * 3] + T[2];
}

__global__
void Error(int n, float* aux, float* D, float* M, int* idx, float* error, int iteration)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	aux[0 + i * 3] = pow(M[0 + idx[i] * 3] - D[0 + i * 3], 2);
	aux[1 + i * 3] = pow(M[1 + idx[i] * 3] - D[1 + i * 3], 2);
	aux[2 + i * 3] = pow(M[2 + idx[i] * 3] - D[2 + i * 3], 2);
	__syncthreads();

	for (int s = 1; s < n; s *= 2)//parallel reduction
	{
		if (i % (2 * s) == 0)
		{
			aux[0 + i * 3] += aux[0 + (i + s) * 3];
			aux[1 + i * 3] += aux[1 + (i + s) * 3];
			aux[2 + i * 3] += aux[2 + (i + s) * 3];
		}
		__syncthreads();
	}

	if (i == 0)
	{
		error[iteration] = (float)sqrt((aux[0] + aux[1] + aux[2]) / (float)n);
		//printf("Error: %f\n",error[iteration]);
	}
}


int main(void)
{
	int num_points, i, j, k;
	float ti[3], ri[3];
	float lin_space[WIDTH], lenght;

	num_points = WIDTH * WIDTH;//number of points
	lenght = XY_max - XY_min;

	//for this specific case the number of points of the 2 clouds are the same
	int d_points = num_points;
	int m_points = num_points;

	////////////////1st:Creation of the synthetic data//////////////

	//create an array with all points equally separated
	int n = WIDTH;
	for (i = 0; i < WIDTH; i++)
	{
		lin_space[i] = (float)XY_min + ((float)i * (float)lenght) / (float(n) - 1.0f);
	}

	//create the meshgrid
	float* mesh_x = (float*)malloc(num_points * sizeof(float));
	float* mesh_y = (float*)malloc(num_points * sizeof(float));

	if ((mesh_x != NULL) && (mesh_y != NULL))
	{
		i = 0;
		k = 0;
		while (i < num_points)
		{
			j = 0;
			while (j < WIDTH)
			{
				mesh_y[i] = lin_space[j];
				mesh_x[i] = lin_space[k];
				i++; j++;
			}
			k++;
		}
	}
	else return 0;

	//Create the function z = f(x,y) = x^2-y^2
	float* z = (float*)malloc(num_points * sizeof(float));
	for (i = 0; i < num_points; i++) z[i] = pow(mesh_x[i], 2) - pow(mesh_y[i], 2);

	//Create data point cloud matrix
	size_t bytesD = (size_t)d_points * (size_t)3 * sizeof(float);
	float* h_D = (float*)malloc(bytesD);

	k = 0;
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			if (j == 0) h_D[k] = mesh_x[i];
			if (j == 1) h_D[k] = mesh_y[i];
			if (j == 2) h_D[k] = z[i];
			k++;
		}
	}

	//printf("Data point cloud\n");
	//printScloud(h_D, num_points, num_points);

	//Translation values
	ti[0] = 1.0f;//x
	ti[1] = -0.3f;//y
	ti[2] = 0.2f;//z

	//Rotation values (rad)
	ri[0] = 1.0f;//axis x
	ri[1] = -0.5f;//axis y
	ri[2] = 0.05f;//axis z

	float h_r[9] = {};
	float cx = (float)cos(ri[0]); float cy = (float)cos(ri[1]); float cz = (float)cos(ri[2]);
	float sx = (float)sin(ri[0]); float sy = (float)sin(ri[1]); float sz = (float)sin(ri[2]);
	h_r[0] = cy * cz; h_r[1] = (cz * sx * sy) + (cx * sz); h_r[2] = -(cx * cz * sy) + (sx * sz);
	h_r[3] = -cy * sz; h_r[4] = (cx * cz) - (sx * sy * sz); h_r[5] = (cx * sy * sz) + (cz * sx);
	h_r[6] = sy; h_r[7] = -cy * sx; h_r[8] = cx * cy;
	//printf("Ri:\n");
	//printScloud(h_r,3,3);

	//Create model point cloud matrix (target point cloud)
	//every matrix is defined using the colum-major order
	size_t bytesM = (size_t)m_points * (size_t)3 * sizeof(float);
	float* h_M = (float*)malloc(bytesM);

	//h_M = h_r*h_D
	SmatrixMul(h_r, h_D, h_M, 3, num_points, 3);
	//h_M = h_M + t
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			h_M[j + i * 3] += ti[j];
		}
	}
	//printf("\nModel point cloud\n");
	//printScloud(h_M, NUM_POINTS, NUM_POINTS);

	/////////End of 1st/////////

	//since this lines 
	//p assumes the value of D
	//q assumes the value of M
	//number of p and q points
	int p_points = d_points;
	int q_points = m_points;
	float* d_p, * d_q;
	hipMalloc(&d_p, bytesD);//p points cloud
	hipMalloc(&d_q, bytesM);//p points cloud//q point cloud
	//transfer data from D and M to p and q
	hipMemcpy(d_p, h_D, bytesD, hipMemcpyHostToDevice);//copy data cloud to p
	hipMemcpy(d_q, h_M, bytesM, hipMemcpyHostToDevice);//copy model cloud to q
	hipError_t err = hipSuccess;//for checking errors in kernels
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/////////2nd: Normals estimation/////////
	int GridSize = 8;
	int BlockSize = q_points / GridSize;
	printf("For normals:\nGrid Size: %d, Block Size: %d\n", GridSize, BlockSize);

	int* d_NeighborIds = NULL;
	float* d_dist = NULL;
	size_t neighbors_size = (size_t)p_points * (size_t)q_points * sizeof(int);
	hipMalloc(&d_NeighborIds, neighbors_size);
	hipMalloc(&d_dist, (size_t)p_points * (size_t)q_points * sizeof(float));
	k = 4;//number of nearest neighbors

	float* d_normals;
	hipMalloc(&d_normals, 3 * (size_t)q_points * sizeof(float));

	hipEventRecord(start);//start time normals estimation
	knn << < GridSize, BlockSize >> > (d_q, q_points, d_q, q_points, d_NeighborIds, k + 1, d_dist);
	if (err != hipSuccess)
		printf("Error in knn kernel: %s\n", hipGetErrorString(err));
	hipDeviceSynchronize();
	/*int* h_NeighborIds = (int*)malloc(neighbors_size);
	hipMemcpy(h_NeighborIds, d_NeighborIds, neighbors_size, hipMemcpyDeviceToHost);
	printf("Neighbor IDs:\n");
	for (i = 0; i < p_points; i++)
	{
		printf("%d: ", i + 1);
		for (j = 0; j < k + 1; j++) printf("%d ", h_NeighborIds[j + i * q_points] + 1);
		printf("\n");
	}
	printf("\n");*/
	Normals << < GridSize, BlockSize >> > (d_q, d_NeighborIds, p_points, q_points, k, d_normals);
	if (err != hipSuccess)
		printf("Error in normals kernel: %s\n", hipGetErrorString(err));
	hipDeviceSynchronize();
	/*float* h_normals = (float*)malloc(bytesM);
	hipMemcpy(h_normals, d_normals, bytesM, hipMemcpyDeviceToHost);
	printf("Normals:\n");
	for (i = 0; i < q_points; i++)
	{
		printf("%d: ", i + 1);
		for (j = 0; j < 3; j++) printf("%.3f ", h_normals[j + i * 3]);
		printf("\n");
	}
	printf("\n");*/

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);
	printf("Normals were calculated in %f ms\n\n", miliseconds);

	/////////End of 2nd/////////

	/////////3rd: ICP algorithm/////////

	int iteration = 0;
	GridSize = 8;
	BlockSize = p_points / GridSize;
	printf("For ICP loop:\nGrid Size: %d, Block Size: %d\n", GridSize, BlockSize);

	float* d_aux = NULL;
	hipMalloc(&d_aux, bytesD);

	int* d_idx = NULL;//index vector (used for correspondence)
	hipMalloc(&d_idx, (size_t)p_points * sizeof(int));
	//int* h_idx = (int*)malloc(D_size * sizeof(int));

	float* d_temp_r = NULL;//temporary rotation matrix
	float* d_temp_T = NULL;//temporary transalation vector
	hipMalloc(&d_temp_r, sizeof(float) * 9);
	hipMalloc(&d_temp_T, sizeof(float) * 3);
	float* h_temp_r = (float*)malloc(sizeof(float) * 9);
	float* h_temp_T = (float*)malloc(sizeof(float) * 3);

	float* d_error = NULL;
	hipMalloc(&d_error, (size_t)(MAX_ITER + 1) * sizeof(float));
	hipMemset(d_error, 0, (size_t)(MAX_ITER + 1) * sizeof(float));
	//float* h_error = (float*)malloc((size_t)(MAX_ITER + 1) * sizeof(float));

	float* d_C = NULL, * d_b = NULL;//for the system of linear equations (minimization)
	hipMalloc(&d_C, 36 * sizeof(float));
	hipMalloc(&d_b, 6 * sizeof(float));
	float* h_b = (float*)malloc(6 * sizeof(float));
	float* d_cn, * d_C_total, * d_b_total;
	hipMalloc(&d_cn, 6 * (size_t)p_points * sizeof(float));
	hipMalloc(&d_C_total, 21 * (size_t)p_points * sizeof(float));
	hipMalloc(&d_b_total, 6 * (size_t)p_points * sizeof(float));

	//cuBLAS handle
	hipblasHandle_t cublasH;
	hipblasCreate(&cublasH);

	//cuSolver handle
	hipsolverHandle_t cusolverH;
	hipsolverDnCreate(&cusolverH);
	int Lwork = 0;
	float* d_work = NULL;
	int* devInfo = NULL;
	hipMalloc(&devInfo, sizeof(int));

	hipEventRecord(start);
	while (iteration < MAX_ITER)
	{
		//////////////////Matching step/////////////////
		Matching << <GridSize, BlockSize >> > (d_p, d_q, q_points, d_idx);
		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error in matching kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();
		/*hipMemcpy(h_idx, d_idx, D_size*sizeof(int), hipMemcpyDeviceToHost);
		printf("Index values:\n");
		printIarray(h_idx, D_size);*/
		/////////////////end of Matching/////////////////

		/////////////////Minimization step (point-to-plane)/////////////////

		Cxb << < GridSize, BlockSize >> > (d_p, p_points, d_q, q_points, d_idx, d_normals, d_cn, d_C_total, d_b_total, d_C, d_b);
		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error in Cxb kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();

		//Allocate the buffer
		hipsolverDnSpotrf_bufferSize(cusolverH, HIPBLAS_FILL_MODE_UPPER, 6, d_C, 6, &Lwork);
		hipMalloc(&d_work, sizeof(float) * Lwork);//allocate memory for the buffer
		//Find the triangular Cholesky factor
		hipsolverDnSpotrf(cusolverH, HIPBLAS_FILL_MODE_UPPER, 6, d_C, 6, d_work, Lwork, devInfo);
		//solve the system of linear equations
		hipsolverDnSpotrs(cusolverH, HIPBLAS_FILL_MODE_UPPER, 6, 1, d_C, 6, d_b, 1, devInfo);//d_b holds the answer
		hipMemcpy(h_b, d_b, 6 * sizeof(float), hipMemcpyDeviceToHost);//move b to the CPU

		//rotation matrix
		cx = (float)cos(h_b[0]); cy = (float)cos(h_b[1]); cz = (float)cos(h_b[2]);
		sx = (float)sin(h_b[0]); sy = (float)sin(h_b[1]); sz = (float)sin(h_b[2]);
		h_temp_r[0] = cy * cz; h_temp_r[3] = cz * sx * sy - cx * sz;  h_temp_r[6] = cx * cz * sy + sx * sz;
		h_temp_r[1] = cy * sz; h_temp_r[4] = cx * cz + sx * sy * sz; h_temp_r[7] = cx * sy * sz - cz * sx;
		h_temp_r[2] = -sy; h_temp_r[5] = cy * sx; h_temp_r[8] = cx * cy;
		//translation vector
		h_temp_T[0] = h_b[3];
		h_temp_T[1] = h_b[4];
		h_temp_T[2] = h_b[5];

		hipMemcpy(d_temp_r, h_temp_r, 9 * sizeof(float), hipMemcpyHostToDevice);//move temp_r to the GPU
		hipMemcpy(d_temp_T, h_temp_T, 3 * sizeof(float), hipMemcpyHostToDevice);//move temp_T to the GPU
		/////////////////end of Minimization/////////////////

		/////////////////Transformation step/////////////////
		RyT << <GridSize, BlockSize >> > (d_temp_r, d_temp_T, d_p, d_aux);
		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error in RyT kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();
		hipblasScopy(cublasH, 3 * p_points, d_aux, 1, d_p, 1);
		/////////////////end of Transformation/////////////////

		/////////////////Error estimation/////////////////
		hipMemset(d_aux, 0, bytesD);
		Error << <GridSize, BlockSize >> > (n, d_aux, d_p, d_q, d_idx, d_error, iteration);
		err = hipGetLastError();
		if (err != hipSuccess)
			printf("Error in error kernel: %s\n", hipGetErrorString(err));
		hipDeviceSynchronize();
		/////////////////end of Error estimation/////////////////

		iteration++;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	miliseconds = 0.0f;
	hipEventElapsedTime(&miliseconds, start, stop);
	printf("Elapsed time: %f ms\n", miliseconds);
	/////////End of 3rd/////////

	free(mesh_x), free(mesh_y), free(z);
	free(h_M), free(h_D);
	hipFree(d_p), hipFree(d_q), hipFree(d_aux);
	hipFree(d_NeighborIds), hipFree(d_dist);//free(h_NeighborIds);
	hipFree(d_normals);//free(h_normals)
	hipFree(d_idx); //free(h_idx);
	hipFree(d_error);
	hipFree(d_work), hipFree(devInfo);
	hipFree(d_C), hipFree(d_b), free(h_b);
	hipFree(d_cn), hipFree(d_C_total), hipFree(d_b_total);
	hipFree(d_temp_r), hipFree(d_temp_T), free(h_temp_r), free(h_temp_T);

	return 0;
}

//double matrix multiplication colum-major order
void SmatrixMul(float* A, float* B, float* C, int m, int n, int k)
{
	int i, j, q;
	float temp = 0.0f;
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < m; j++)
		{
			temp = 0.0f;
			for (q = 0; q < k; q++) temp += A[j + q * m] * B[q + i * k];
			C[j + i * m] = temp;
		}
	}
}

//print matrix
void printScloud(float* cloud, int num_points, int points2show)
{
	int i, j, offset;
	printf("x\ty\tz\n");
	if (points2show <= num_points)
	{
		for (i = 0; i < points2show; i++)
		{
			for (j = 0; j < 3; j++)
			{
				offset = j + i * 3;
				printf("%.4f\t", cloud[offset]);
				if (j % 3 == 2) printf("\n");
			}
		}
	}
	else printf("The cloud can't be printed\n\n");
}

//print vector with double values
void printSarray(float* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%.4f ", array[i]);
	}
	printf("\n");
}

//print vector with integer values
void printIarray(int* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%d ", array[i]);
	}
	printf("\n");
}
